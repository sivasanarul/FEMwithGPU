#include <stdio.h>
#include<iostream>
#include <hip/hip_runtime.h>
#include <time.h>
#include <sys/time.h>

using namespace std;
__global__ void matvec(float *vec, float *mat, float *out, const int N, const int M){
    int tid=threadIdx.x+blockIdx.x*blockDim.x;
        float sum=0;
    
        for(int i=0; i<N; i++)
            sum += vec[i]*mat[(tid*M)+i];
        out[tid]=sum;    
}

extern double mysecond();
void init_array(float *a, const int N);
void init_mat(float *a, const int N, const int M);
void print_array(float *a, const int N, char *d);
void print_mat(float *a, const int N, const int M, char *d);

int main (void) {

    float *a, *b, *c, *d;
    float *dev_a, *dev_b, *dev_c;
    
    double t;
    int N= 32768;
    int M=N;
    
    // Allocate host memory 
    a=(float*)malloc(sizeof(float)*N);
    b=(float*)malloc(sizeof(float)*N*M);
    c=(float*)malloc(sizeof(float)*M);
    d=(float*)malloc(sizeof(float)*M);

    // Initialize matrices    
    init_array(a, N);
    init_mat(b, N, M);
    init_array(c, M);

    // Allocate device memory
    hipMalloc((void**)&dev_a, sizeof(float)*N);
    hipMalloc((void**)&dev_b, sizeof(float)*N*M);
    hipMalloc((void**)&dev_c, sizeof(float)*M);

    int blocksize = 32; // value usually chosen by tuning and hardware constraints
    int nblocks = N / blocksize;
    cout<<"\nblocksize         :  "<<((double)blocksize);
    cout<<"\nnumber of blocks  :  "<<((double)nblocks);
 
    t = mysecond();
    hipMemcpy(dev_a, a, sizeof(float)*N,   hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, sizeof(float)*N*M, hipMemcpyHostToDevice);
    t = (mysecond() - t);
    printf ("\nElapsed time for copy from host to device   = %g\n", t );

    t = mysecond();
    // matrix vector product    
    matvec<<<nblocks, blocksize>>>(dev_a, dev_b, dev_c, N, M);
    t = (mysecond() - t);
    printf ("\nElapsed time for matrix vector product in n block = %g\n", t );

    t = mysecond();
    // Transfer data from device to host memory 
    hipMemcpy(c, dev_c, sizeof(float)*M, hipMemcpyDeviceToHost);
    t = (mysecond() - t);
    printf ("\nElapsed time for copy from device to host   = %g\n", t );   
        
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    // verify the kernel implementation      
    float sum=0;
    for(int row=0;row<N;row++)
	    {
		sum=0;
		for(int col=0;col<N;col++)
		{
		      sum=sum+b[row*N+col]*a[col];  
		    
		}
	      d[row]=sum;
	    } 
	    
    float error=0;
    for(int i=0;i<N;i++)
        error+=d[i]-c[i];
     
    
    cout<<"Error: "<<error;       
    cout<<"\n\n"; 

    // Deallocate host memory
    free(a); 
    free(b); 
    free(c);
    free(d);
            
    return 0;
};

void init_array(float *a, const int N) {
        int i;
        for(i=0; i<N; i++)
                a[i] = 1.0;
}
void init_mat(float *a, const int N, const int M) {
        int i, j;
        for(i=0; i<N; i++)
            for(j=0; j<M; j++)
                    a[i*M+j] = 2.0;
}
void print_array(float *a, const int N, char *d) {
        int i;
        for(i=0; i<N; i++)
                printf("\n%s[%d]: %f",d, i, a[i]);
    printf("\n");
}
void print_mat(float *a, const int N, const int M, char *d) {
        int i, j;
        for(i=0; i<N; i++){
        printf("\n%s[%d]:", d, i);
        for (j=0; j<M; j++)
                    printf("\t%6.4f", a[i*M+j]);
    }
    printf("\n");
}

double mysecond()
{
    struct timeval tp;
    struct timezone tzp;
    int i;
    
    i = gettimeofday(&tp,&tzp);
    return ( (double) tp.tv_sec + (double) tp.tv_usec  * 1.e-6);
}
