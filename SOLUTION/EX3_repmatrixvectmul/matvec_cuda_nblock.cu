#include <stdio.h>
#include<iostream>
#include <hip/hip_runtime.h>
#include <time.h>
# include <omp.h>
#include <sys/time.h>

using namespace std;
__global__ void matvec(float *vec, float *mat, float *out, const int N, const int M){
    int tid=threadIdx.x + blockIdx.x*blockDim.x;
        float sum = 0;
    
        for(int i = 0; i < N; i++)
            sum += vec[i]*mat[(tid*M) + i];
        out[tid] = sum;
    
}

extern double mysecond();
void init_array(float *a, const int N);
void init_mat(float *a, const int N, const int M);
void print_array(float *a, const int N, char *d);
void print_mat(float *a, const int N, const int M, char *d);

int main (void) {

    float *a, *b, *c, *d;
    float *dev_a, *dev_b, *dev_c;
    double t;
    
    int N= 32768;
    int M=N;
    
    // Allocate host memory
    a = (float*)malloc(sizeof(float)*N);
    b = (float*)malloc(sizeof(float)*N*M);
    c = (float*)malloc(sizeof(float)*M);
    d = (float*)malloc(sizeof(float)*M);
    
    init_array(a, N);
    init_mat(b, N, M);
    init_array(c, M);

    // Allocate device memory
    hipMalloc((void**)&dev_a, sizeof(float)*N);
    hipMalloc((void**)&dev_b, sizeof(float)*N*M);
    hipMalloc((void**)&dev_c, sizeof(float)*M);

    int blocksize = 256; // value usually chosen by tuning and hardware constraints
    int nblocks = N / blocksize;
    
    printf("\n\nRunning Kernel...\n\n");
    
    t = mysecond();  
    // Transfer data from host to device memory  
    hipMemcpy(dev_b, b, sizeof(float)*N*M, hipMemcpyHostToDevice);
    hipMemcpy(dev_a, a, sizeof(float)*N,   hipMemcpyHostToDevice);
    t = (mysecond() - t);
    printf ("\nElapsed seconds for data transfer from Host to Device = %g\n", t);
        
    t = mysecond();     
    // matrix vector product for 100 iterations   
    int iter; int niter; niter = 500;
    for (iter = 0; iter < niter; iter++)
            {  
            // matrix vector product  	    	    
	    matvec<<<nblocks, blocksize>>>(dev_a, dev_b, dev_c, N, M);
	    hipDeviceSynchronize();
	    }
    t = (mysecond() - t);
    printf ("\nElapsed seconds for executing kernel = %g\n", t);
  
    
    t = mysecond();
    // Transfer data from device to host memory     
    hipMemcpy(c, dev_c, sizeof(float)*M, hipMemcpyDeviceToHost);    
    t = (mysecond() - t);
    printf ("\nElapsed seconds for data transfer from Device to Host = %g\n", t); 
    
    // Deallocate device memory           
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

       
    float sum = 0;
    for(int row = 0; row < N; row++)
	    {
		sum = 0;
		for(int col = 0; col < N; col++)
		{
		      sum=sum + b[row*N + col]*a[col];  
		    
		}
	      d[row] = sum;
	    } 
	    
    float error = 0;
    for(int i = 0; i < N; i++)
        error += d[i] - c[i];
     
    printf ("\nError   = %g\n", error );    
     
    // Deallocate host memory
    free(a); 
    free(b); 
    free(c);
    free(d); 
    return 0;
};

void init_array(float *a, const int N) {
        int i;
        for(i=0; i<N; i++)
                a[i] = 1.0;
}
void init_mat(float *a, const int N, const int M) {
        int i, j;
        for(i=0; i<N; i++)
            for(j=0; j<M; j++)
                    a[i*M+j] = 2.0;
}
void print_array(float *a, const int N, char *d) {
        int i;
        for(i=0; i<N; i++)
                printf("\n%s[%d]: %f",d, i, a[i]);
    printf("\n");
}
void print_mat(float *a, const int N, const int M, char *d) {
        int i, j;
        for(i=0; i<N; i++){
        printf("\n%s[%d]:", d, i);
        for (j=0; j<M; j++)
                    printf("\t%6.4f", a[i*M+j]);
    }
    printf("\n");
}

double mysecond()
{
    struct timeval tp;
    struct timezone tzp;
    gettimeofday(&tp,&tzp);
    return ( (double) tp.tv_sec + (double) tp.tv_usec  * 1.e-6);
}
