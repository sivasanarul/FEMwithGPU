#include <stdio.h>
#include <time.h>
#include <math.h>
#include <assert.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <sys/time.h>

#define array_size 268435456

__global__ void vector_add(float *out, float *a, float *b, int n){
     int index = blockIdx.x *blockDim.x + threadIdx.x; 
     if (index < n){
     out[index] = a[index] + b[index];}
}

extern double mysecond();

int main(){
    float *a, *b, *out;
    float *d_a, *d_b, *d_out;
    double t;
    
    // Allocate host memory    
    a   = (float*)malloc(sizeof(float) * array_size); 
    b   = (float*)malloc(sizeof(float) * array_size);
    out = (float*)malloc(sizeof(float) * array_size);

    // Initialize array
    for(int i = 0; i < array_size; i++){
        a[i] = 1.0f; 
        b[i] = 2.0f;
    }
    
    // Allocate device memory
    hipMalloc((void**)&d_a, sizeof(float)*array_size);
    hipMalloc((void**)&d_b, sizeof(float)*array_size);
    hipMalloc((void**)&d_out, sizeof(float)*array_size);
 
    t = mysecond();
    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float)*array_size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float)*array_size, hipMemcpyHostToDevice);
    t = (mysecond() - t);
    printf ("\nElapsed time for copy from host to device   = %g\n", t );
    
    int block_size = 256;
    int grid_size  = (array_size + block_size) / block_size; 
    t = mysecond();
    // Vector addition    
    vector_add<<<grid_size, block_size>>>(d_out, d_a, d_b, array_size);
    t = (mysecond() - t);
    printf ("\nElapsed time for vector addition in n blocks = %g\n", t ); 
    
    t = mysecond();
    // Transfer data from device to host memory
    hipMemcpy(out, d_out, sizeof(float)*array_size, hipMemcpyDeviceToHost);
    t = (mysecond() - t);
    printf ("\nElapsed time for copy from device to host   = %g\n", t );
    
    // Deallocate device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    // Deallocate host memory
    free(a); 
    free(b); 
    free(out);

}

double mysecond()
{
    struct timeval tp;
    struct timezone tzp;
    int i;
    
    i = gettimeofday(&tp,&tzp);
    return ( (double) tp.tv_sec + (double) tp.tv_usec  * 1.e-6);
}
